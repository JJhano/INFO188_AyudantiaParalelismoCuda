#include <cstdio>
#include <cstdlib>
// #include <omp.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
using namespace std;
// Tamaño de bloque
#define BSIZE 32
#define BSIZE1D 1024
#define PRINT_SIZE 50

__global__ void gpu_sim(bool *M1, int n, bool *M2);
__device__ int count_live_neighbour_cell_GPU(bool *M, int n, int tidx, int tidy);

void inicializa_matrix(bool *M1, int n, int seed);
void printM(bool *M1, int n);
void copy(bool *M1, int n, bool *M2);
void cpu_sim(bool *M1, int n, bool *M2);
void set_gpu(int gpu_id);


int main(int argc, char **argv){
    // Se 
    if(argc != 6){
        fprintf(stderr, "run as ./prog <gpu-id>  n seed pasos <block-size>\n");
        exit(EXIT_FAILURE);
    }
    // Inicializacion 
    printf("Inicializando.....\n");
    string str = "";
    int gpu_id = atoi(argv[1]); // ID de la gpu
    int n = atoi(argv[2]); // Size problem [nxN]
    int seed = atoi(argv[3]); // Seed 
    int steps = atoi(argv[4]); // Steps number
    int nb = atoi(argv[5]);
    printf("<gpu_id: %d>, <n: %d> , <seed: %d> , <steps: %d>, <nb: %d>\n", gpu_id, n, seed, steps, nb);
    bool * M1 = new bool[n*n];
    bool * M2 = new bool[n*n];
    inicializa_matrix(M1, n, seed);
    if( n <= PRINT_SIZE ) {
        printf("Matrix inicial: \n");
        printM(M1,n);
    }
    set_gpu(gpu_id);
    printf("Inicializando variables para GPU\n");
    float msecs = 0.0f;
    bool * dx, * dy;
    // Se ingresan las matrices en la memoria de la GPU 
    hipMalloc(&dx, sizeof(bool) * (n * n));
    hipMalloc(&dy, sizeof(bool) * (n * n));
    hipMemcpy(dx, M1, sizeof(bool) * (n * n), hipMemcpyHostToDevice);
    // Se crean los eventos para medir el tiempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Se eligen los tamaños de block y grid 
    hipEventRecord(start);
    dim3 block(BSIZE, BSIZE, 1);
    dim3 grid ((n + (BSIZE - 1))/BSIZE, (n + (BSIZE - 1))/BSIZE, 1);
    if(nb != 0){
        block = dim3(BSIZE, 1, 1);
        grid = dim3(nb, 1, 1);
    }
    while(str == "" && steps != 0){
        gpu_sim<<<grid, block>>>(dx, n, dy);
        if(n <= PRINT_SIZE){
            hipMemcpy(M2, dy, sizeof(bool)*(n*n), hipMemcpyDeviceToHost);
            printf("------------\n");
            printM(M2,n);
            cout << "Para hacer otra generacion <enter>, para terminar cualquier cosa\n>";
            getline(cin, str);
        }
        // Se sincroniza
        hipDeviceSynchronize();
        hipMemcpy(dx, dy, sizeof(bool) * (n * n), hipMemcpyDeviceToDevice);
        steps--;
    }    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    // Se calcula el tiempo
    hipEventElapsedTime(&msecs, start, stop);
    //Se traspasa la informacion al host
    hipMemcpy(M2, dy, sizeof(bool)*(n*n), hipMemcpyDeviceToHost);
    printf("done: time gpu: %f secs\n", msecs/1000.0f);
    //Se libera la memoria de la GPU
    hipFree(dx);
    hipFree(dy);
    
    // Se libera la memoria utilizada
    delete M1;
    delete M2;
    printf("El programa termino con exito!\n");
    fflush(stdout);
    exit(EXIT_SUCCESS);
}


/* Funciones auxiliares */


void set_gpu(int gpu_id){
    // Se selecciona la gpu y se muestra
    hipSetDevice(gpu_id);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, gpu_id);
    printf("GPU: %s\n", prop.name);
}

void copy(bool *M1, int n, bool *M2){
    for (int i = 0; i < n * n; i++){
            M1[i] = M2[i];
    }
}

void inicializa_matrix(bool *M1, int n, int seed){
    srand(seed);
    for(int i = 0; i < n * n; ++i){
        M1[i] = rand()%2;
    }
}

void printM(bool *M1, int n){
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            if(M1[i * n + j]) printf("1 ");
            else printf("0 ");
        }
        printf("\n");
    }
}
/* Funcion para ejecutar en GPU */
// Cuenta la cantidad de vecinos vivos que está alrededor de la celula
__device__ int count_live_neighbour_cell_GPU(bool *M, int n, int tidx, int tidy){
    int neighbour_live_cell = 0;
    for (int i = tidx - 1; i <= tidx + 1; i++) {
        for (int j = tidy - 1; j <= tidy + 1; j++) {
            if ((i == tidx && j == tidy) || (i < 0 || j < 0)
                || (i >= n || j >= n)) continue;
            if (M[j*n + i]) neighbour_live_cell++;
        }
    }
    return neighbour_live_cell;
}
    /* Kernel principal*/
// Cambia los valores de la matriz segun las reglas del juego de la vida
__global__ void gpu_sim(bool *M1, int n, bool *M2){
    // Numero de celulas vivas alrededor
    int neighbour_live_cell = 0;
    int tidx = blockIdx.x * blockDim.x  +  threadIdx.x;
    int tidy = blockIdx.y * blockDim.y +  threadIdx.y;
    int pos = (tidy * n) + tidx;
    if(tidx < n && tidy < n){
        neighbour_live_cell = count_live_neighbour_cell_GPU(M1,n,tidx,tidy);
        if (M1[pos] && (neighbour_live_cell == 2 
            || neighbour_live_cell == 3)) M2[pos] = 1;
        else if (!(M1[pos]) && neighbour_live_cell == 3) M2[pos] = 1;
        else M2[pos] = 0;
    }

}